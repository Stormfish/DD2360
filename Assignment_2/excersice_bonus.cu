#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <math.h>
#include "utils.h"

constexpr int TRIALS_PER_THREAD = 100000;
constexpr int THREADS_PER_BLOCK = 256;
constexpr int BLOCKS_PER_KERNEL = 64;
constexpr int TOTAL_THREADS		= THREADS_PER_BLOCK*BLOCKS_PER_KERNEL; 
constexpr int TOTAL_ITERATIONS  = THREADS_PER_BLOCK*BLOCKS_PER_KERNEL*TRIALS_PER_THREAD; 

struct Particle{
	float3 pos; 
	float3 vel; 
};

__global__ void gpu_random(hiprandState *states, float* res) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid >= TOTAL_THREADS){ 
    	return; 
    } 

    float x;
    float y; 
    int seed = 0+tid; 

	hiprand_init(seed, tid, 0, &states[tid]);  // 	Initialize CURAND
	for(int i = 0; i < TRIALS_PER_THREAD; i++) {
		x = hiprand_uniform (&states[tid]);
		y = hiprand_uniform (&states[tid]);
		//x²+y²=<r²?
		if(x*x+y*y<1.0){
			res[tid]+=1.0; 
		} 
		//printf("tid:%d %f,%f, %f\n",tid, x, y, res[tid]);
	}	

}


int main(){
	//generate random velocities & positions
	hiprandState *dev_random;
	hipMalloc((void**)&dev_random, TOTAL_THREADS*sizeof(hiprandState));
	float* d_hits;
	hipMalloc((void**)&d_hits, 	TOTAL_THREADS*sizeof(hiprandState));

	double iStart = cpuSecond();
	gpu_random<<<BLOCKS_PER_KERNEL, THREADS_PER_BLOCK>>>(dev_random, d_hits); 
	hipDeviceSynchronize();
	double iElaps = cpuSecond() - iStart;
	printf("Done! in:%fs \n", iElaps); 


	std::vector<float> results(TOTAL_THREADS);
	hipMemcpy(results.data(), (void*)d_hits, results.size()*sizeof(float), hipMemcpyDeviceToHost); 

	float sum=0; 
	for (auto& f:results)
	{
		sum += f; 
	}

	float pi = 4*sum/TOTAL_ITERATIONS; 
	printf("sum:%f\n", sum); 
	printf("pi=%f\n", pi);
	return 0; 
}