#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "utils.h"


constexpr int ARRAY_SIZE = 100000; 
constexpr int num_threads = 256; 
constexpr int num_blocks = ARRAY_SIZE/num_threads + 1; 

__global__ 
void saxpy(float* x, float* y, float c){ 
    const unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;  
    if(tid >= ARRAY_SIZE){ 
    	return; 
    }
    y[tid] = x[tid] + c; 
}

void saxpy_cpu(float* x, float* y, float c){
    for(int i=0; i<ARRAY_SIZE; ++i){ 
    	y[i] = x[i] + c; 
    }
}

int main(){

	std::vector<float> v(ARRAY_SIZE, 5.0); 
	float* x_array; 
	float* y_array; 

	hipMalloc( (void **)&x_array, v.size()*sizeof(float) ); 
	hipMalloc( (void **)&y_array, v.size()*sizeof(float) ); 
	hipMemcpy( (void*)x_array, v.data(), v.size()*sizeof(float), hipMemcpyHostToDevice ); 

	printf("Computing SAXPY on the GPU…:"); 
	double iStart = cpuSecond();
	saxpy<<<num_blocks, num_threads>>>(x_array, y_array, 5); 
	hipDeviceSynchronize();
	double iElaps = cpuSecond() - iStart;
	hipMemcpy( v.data(), (void*)y_array, v.size()*sizeof(float), hipMemcpyDeviceToHost );	
	printf("Done! in:%fs \n", iElaps); 

	std::vector<float> v2(ARRAY_SIZE, 5.0); 
	printf("Computing SAXPY on the CPU…:"); 
	iStart = cpuSecond();
	saxpy_cpu(v2.data(), v2.data(), 5); 
	iElaps = cpuSecond() - iStart;
	printf("Done! in:%fs \n", iElaps); 

	float diff=0.0; 
	for(int i=0; i<ARRAY_SIZE; ++i){
		diff += v[i]-v2[i]; 
	} 
	printf("\nsum of differences: %f\n", diff); 

	


	return 0; 
}

